#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <random>

//#include "HashGridProcessor.h"

// thrust for sorting
#include "thrust/device_ptr.h"
#include "thrust/sort.h"
//#include <hip/hip_runtime.h>

#include "Loader.h"
#define N 1

#define HANDLE_ERROR(err)\
	(handleCudaError(err, __FILE__, __LINE__))


static void handleCudaError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

__host__ __device__ int3 operator+(int3 a, int3 b)
{
	return make_int3(a.x + b.x, a.y + b.y, a.z + b.z);
}

// calculate position grid
__host__ __device__ int3 calcGridPos(float3 p, PSystemInfo pSysInfo)
{
	int3 gridPos;
	gridPos.x = floor((p.x - pSysInfo.worldOrigin.x) / pSysInfo.cellSize.x);
	gridPos.y = floor((p.y - pSysInfo.worldOrigin.y) / pSysInfo.cellSize.y);
	gridPos.z = floor((p.z - pSysInfo.worldOrigin.z) / pSysInfo.cellSize.z);
	return gridPos;
}

__device__ float3 calcGridPosFloat(float3 p, PSystemInfo pSysInfo)
{
	float3 gridPos;
	gridPos.x = (p.x - pSysInfo.worldOrigin.x) / pSysInfo.cellSize.x;
	gridPos.y = (p.y - pSysInfo.worldOrigin.y) / pSysInfo.cellSize.y;
	gridPos.z = (p.z - pSysInfo.worldOrigin.z) / pSysInfo.cellSize.z;
	return gridPos;
}

// calculate hash value in grid
__host__ __device__ uint calcGridHash(int3 gridPos, PSystemInfo pSysInfo)
{
	return gridPos.z * pSysInfo.gridSize.y * pSysInfo.gridSize.x + gridPos.y * pSysInfo.gridSize.x + gridPos.x;
}

// loads the value of type T from char*
template<typename T>
__device__ T load(char* d_begin) {
	const uint size = sizeof(T);
	char values[size];
	for (uint i = 0; i < size; i++) {
		values[i] = d_begin[i];
	}
	return *(T*)values;
}

// saves the value of type T to char*
template<typename T>
__device__ void save(T value, char* d_begin) {
	const uint size = sizeof(T);
	char* values;
	values = (char*)&value;
	for (uint i = 0; i < size; i++) {
		d_begin[i] = values[i];
	}
}

__device__ bool checkCollision(float3 p1, float3 size1, float3 p2, float3 size2) {
	if (abs(p1.x - p2.x) < size1.x + size2.x) {
		if (abs(p1.y - p2.y) < size1.y + size2.y) {
			if (abs(p1.z - p2.z) < size1.z + size2.z) {
				return true;
			}
		}
	}
	return false;
}

__global__ void fillHashGrid(char* d_List, uint* d_HashList, uint* d_IdList, ParticleInfo pInfo, PSystemInfo pSysInfo, bool aligned) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < pInfo.groupCount) {
		float3 f;
		if(aligned)
			f = *(float3*)(d_List + idx * pInfo.stride);
		else {
			f = load<float3>(d_List + idx * pInfo.stride);
		}
			
		
		int3 gridPos = calcGridPos(f, pSysInfo);
		d_HashList[idx] = calcGridHash(gridPos, pSysInfo);
		d_IdList[idx] = idx;
	}
}

__global__ void setCellPointers(uint* d_HashList, uint* d_CellBegin, uint* d_CellEnd, ParticleInfo pInfo) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint hash = d_HashList[idx];
	if (idx < pInfo.groupCount) {

		if (idx == 0)
		{
			d_CellBegin[hash] = 0;
			return;
			
		}
		uint prevHash = d_HashList[idx - 1];
		if (hash != prevHash) {
			d_CellEnd[prevHash] = idx;
			d_CellBegin[hash] = idx;
		}
		if (idx == pInfo.groupCount - 1) {
			d_CellEnd[hash] = idx + 1;
		}
	}
}

void writeToFile(const std::string& name, uint* cellBegin, uint* cellEnd, uint3 gridSize) {
	std::ofstream outputFile(name, std::ifstream::binary);
	for (int i = 0; i < gridSize.x * gridSize.y * gridSize.z; i++) {
		uint64_t num = cellEnd[i] - cellBegin[i];
		outputFile.write((char*) &num, 4);
	}
	outputFile.close();
}

void writeToFile(const std::string& name, std::vector<std::vector<uint>> grid, uint3 gridSize) {
	std::ofstream outputFile(name, std::ifstream::binary);
	for (int i = 0; i < gridSize.x * gridSize.y * gridSize.z; i++) {
		uint64_t num = grid[i].size();
		outputFile.write((char*)&num, 4);
	}
	outputFile.close();
}

ParticleList reduceParticles(const ParticleList pList, float redPercentage) {
	
	std::random_device rd;

	/* Random number generator */
	std::default_random_engine generator(rd());

	/* Distribution on which to apply the generator */
	std::uniform_int_distribution<unsigned int> distribution(0, pList.info.groupCount - 1);

	bool* b = new bool[pList.info.groupCount];
	for (int i = 0; i < pList.info.groupCount; i++) {
		b[i] = false;
	}

	int numRemove = (int)(pList.info.groupCount * redPercentage);
	for (int i = 0; i < numRemove; i++) {
		uint rNum = distribution(generator);
		while (b[rNum] == true) {
			rNum = distribution(generator);
		}
		b[rNum] = true;
	}
	char* output = new char[(pList.info.groupCount - numRemove) * pList.info.stride];
	int index = 0;
	for (int i = 0; i < pList.info.groupCount; i++) {
		if (b[i] == false) {
			for (int a = 0; a < pList.info.stride; a++) {
				output[index * pList.info.stride + a] = pList.data[i * pList.info.stride + a];
			}
			index++;
		}
	}
	ParticleInfo pInfo = pList.info;
	pInfo.groupCount = pList.info.groupCount - numRemove;
	ParticleList particles;
	particles.info = pInfo;
	particles.data = output;
	delete[] b;
	return particles;
}

void benchmarkTimeGPU() {
	int blocksize = 128;
	unsigned long long minTime = -1;
	for (int a = 0; a < 100; a++) {
/*	while(true){
		uint number = 0;
		std::cin >> number;*/
		
		Loader loader("exp2mill.mmpld");

		auto pList = loader.getFrame(20);

		/*
		float3 cellSize;
		cellSize.x = 1;
		cellSize.y = 1;
		cellSize.z = 1;
		PSystemInfo pSysInfo = loader.calcBSystemInfo(cellSize);
		*/
		
		uint3 gridSize;
		gridSize.x = 32;
		gridSize.y = 320;
		gridSize.z = 32;
		PSystemInfo pSysInfo = loader.calcBSystemInfo(gridSize);
		
//		PSystemInfo pSysInfo = loader.calcBSystemInfo(pList);
		

		std::cout << "Grid Size: (" << pSysInfo.gridSize.x << ", " << pSysInfo.gridSize.y << ", " << pSysInfo.gridSize.z << ") Origin: (" << pSysInfo.worldOrigin.x << ", " << pSysInfo.worldOrigin.y << ", " << pSysInfo.worldOrigin.z << ") Cell Size: (" << pSysInfo.cellSize.x << ", " << pSysInfo.cellSize.y << ", " << pSysInfo.cellSize.z << ")" << std::endl;
		
		std::vector<char*> d_List(pList.size());
		std::vector<char*> h_List(pList.size());

		std::vector<uint*> d_HashList(pList.size());
		std::vector<uint*> d_IdList(pList.size());
		std::vector<uint*> d_CellBegin(pList.size());
		std::vector<uint*> d_CellEnd(pList.size());
		
		long long startTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
		for (int n = 0; n < pList.size(); n++) {
			auto p = pList[n];
			
			HANDLE_ERROR(hipMalloc(&d_List[n], p.info.stride * p.info.groupCount));
			HANDLE_ERROR(hipMemcpy(d_List[n], p.data, p.info.stride * p.info.groupCount, hipMemcpyHostToDevice));

			
			HANDLE_ERROR(hipMalloc(&d_HashList[n], sizeof(uint) * p.info.groupCount * N));
//			HANDLE_ERROR(hipMemset(d_HashList[n], -1, sizeof(uint) * p.info.groupCount * N));
			HANDLE_ERROR(hipMalloc(&d_IdList[n], sizeof(uint) * p.info.groupCount * N));
//			HANDLE_ERROR(hipMemset(d_IdList[n], -1, sizeof(uint) * p.info.groupCount * N));
			HANDLE_ERROR(hipMalloc(&d_CellBegin[n], sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));
			HANDLE_ERROR(hipMemset(d_CellBegin[n], -1, sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));
			HANDLE_ERROR(hipMalloc(&d_CellEnd[n], sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));
			HANDLE_ERROR(hipMemset(d_CellEnd[n], -1, sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));

			
			// kernel call
			dim3 dimBlock(blocksize);
			dim3 dimGrid(ceil(pList[n].info.groupCount / (float)blocksize));
			bool isAligned = true;
			if (p.info.stride % 2 != 0)
				isAligned = false;

			
			fillHashGrid << <dimGrid, dimBlock >> > (d_List[n], d_HashList[n], d_IdList[n], p.info, pSysInfo, isAligned);

//			hipDeviceSynchronize();
			
			thrust::sort_by_key(thrust::device_ptr<uint>(d_HashList[n]),
				thrust::device_ptr<uint>(d_HashList[n] + p.info.groupCount * N),
				thrust::device_ptr<uint>(d_IdList[n]));
			
			setCellPointers << <dimGrid, dimBlock >> > (d_HashList[n], d_CellBegin[n], d_CellEnd[n], p.info);
			hipDeviceSynchronize();

//			hipDeviceSynchronize();
			/*
			uint* h_HashList = new uint[p.info.groupCount * N];
			uint* h_IdList = new uint[p.info.groupCount * N];
			HANDLE_ERROR(hipMemcpy(h_HashList, d_HashList[n], sizeof(uint) * p.info.groupCount * N, hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipMemcpy(h_IdList, d_IdList[n], sizeof(uint) * p.info.groupCount * N, hipMemcpyDeviceToHost));

			for (int i = 0; i < p.info.groupCount * N; i++) {
				std::cout << i << ": Hash: " << h_HashList[i] << ", Id: " << h_IdList[i] << std::endl;
			}

			delete[] h_HashList;
			delete[] h_IdList;
			*/
			/*
			uint* h_CellBegin = new uint[pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z];
			uint* h_CellEnd = new uint[pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z];
			HANDLE_ERROR(hipMemcpy(h_CellBegin, d_CellBegin[n], sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z, hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipMemcpy(h_CellEnd, d_CellEnd[n], sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z, hipMemcpyDeviceToHost));
			writeToFile("test.raw", h_CellBegin, h_CellEnd, pSysInfo.gridSize);
			
			uint maxParticle = 0;
			uint maxI = 0;

			for (int i = 0; i < pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z; i++) {
//				std::cout << "Hash: " << i << ", Begin: " << h_CellBegin[i] << ", End: " << h_CellEnd[i] << ", Particles: " << h_CellEnd[i] - h_CellBegin[i] << std::endl;
				if (maxParticle < h_CellEnd[i] - h_CellBegin[i]) {
					maxParticle = h_CellEnd[i] - h_CellBegin[i];
					maxI = i;
				}
					
			}
			std::cout << "Max particle per cube: " << maxParticle << " at " << maxI << std::endl;
			delete[] h_CellBegin;
			delete[] h_CellEnd;
			*/

			/*
			// copy array back for verification
			h_List[n] = new char[p.info.stride * p.info.groupCount];
			HANDLE_ERROR(hipMemcpy(h_List[n], d_List[n], p.info.stride * p.info.groupCount, hipMemcpyDeviceToHost));
			
			// value testing
			for (int i = 0; i < p.info.groupCount; i++) {
				float3* pos = (float3*)(p.data + i * p.info.stride);
				float3* posNew = (float3*)(h_List[n] + i * p.info.stride);
				if (pos->x != posNew->x || pos->y != posNew->y || pos->z != posNew->z)
					std::cout << i << ": " << pos->x << ", " << pos->y << ", " << pos->z << "; " << posNew->x << ", " << posNew->y << ", " << posNew->z << std::endl;
			}
			*/
			
		
		
			std::cout << "New list with " << p.info.groupCount << " particles" << std::endl;
		}
		long long endTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
		std::cout << (endTime - startTime) / 1000.0 << std::endl;

		if (endTime - startTime < minTime)
			minTime = endTime - startTime;
		
		for (int n = 0; n < pList.size(); n++) {
			HANDLE_ERROR(hipFree(d_List[n]));

			HANDLE_ERROR(hipFree(d_HashList[n]));
			HANDLE_ERROR(hipFree(d_IdList[n]));
			HANDLE_ERROR(hipFree(d_CellBegin[n]));
			HANDLE_ERROR(hipFree(d_CellEnd[n]));
			delete[] h_List[n];
			delete[] pList[n].data;
		}


	}

	std::cout << minTime / 1000.0f << std::endl;
}

long long benchmarkPListGPU(ParticleList p, PSystemInfo pSysInfo, int blocksize, int iterations = 1) {
	unsigned long long minTime = -1;
	for (int i = 0; i < iterations; i++) {
		char* d_List;

		uint* d_HashList;
		uint* d_IdList;
		uint* d_CellBegin;
		uint* d_CellEnd;

		
		long long startTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
		HANDLE_ERROR(hipMalloc(&d_List, p.info.stride * p.info.groupCount));
		HANDLE_ERROR(hipMemcpy(d_List, p.data, p.info.stride * p.info.groupCount, hipMemcpyHostToDevice));

		
		HANDLE_ERROR(hipMalloc(&d_HashList, sizeof(uint) * p.info.groupCount * N));

		HANDLE_ERROR(hipMalloc(&d_IdList, sizeof(uint) * p.info.groupCount * N));

		HANDLE_ERROR(hipMalloc(&d_CellBegin, sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));
		HANDLE_ERROR(hipMemset(d_CellBegin, -1, sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));
		HANDLE_ERROR(hipMalloc(&d_CellEnd, sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));
		HANDLE_ERROR(hipMemset(d_CellEnd, -1, sizeof(uint) * pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z));


		// kernel call
		dim3 dimBlock(blocksize);
		dim3 dimGrid(ceil(p.info.groupCount / (float)blocksize));
		bool isAligned = true;
		if (p.info.stride % 2 != 0)
			isAligned = false;

		
		fillHashGrid << <dimGrid, dimBlock >> > (d_List, d_HashList, d_IdList, p.info, pSysInfo, isAligned);


		thrust::sort_by_key(thrust::device_ptr<uint>(d_HashList),
			thrust::device_ptr<uint>(d_HashList + p.info.groupCount * N),
			thrust::device_ptr<uint>(d_IdList));

		setCellPointers << <dimGrid, dimBlock >> > (d_HashList, d_CellBegin, d_CellEnd, p.info);
		hipDeviceSynchronize();

		long long endTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
//		std::cout << (endTime - startTime) / 1000.0 << std::endl;

		if (endTime - startTime < minTime)
			minTime = endTime - startTime;

		HANDLE_ERROR(hipFree(d_List));

		HANDLE_ERROR(hipFree(d_HashList));
		HANDLE_ERROR(hipFree(d_IdList));
		HANDLE_ERROR(hipFree(d_CellBegin));
		HANDLE_ERROR(hipFree(d_CellEnd));
	}
	std::cout << "GPU: Particle count: " << p.info.groupCount << " Time: " << minTime / 1000.0f << std::endl;
	return minTime;
}

long long benchmarkPListCPU(ParticleList p, PSystemInfo pSysInfo, int iterations = 1) {
	unsigned long long minTime = -1;
	for (int a = 0; a < iterations; a++) {

		long long startTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
		std::vector<std::vector<uint>> grid(pSysInfo.gridSize.x * pSysInfo.gridSize.y * pSysInfo.gridSize.z);

		for (uint i = 0; i < p.info.groupCount; i++)
			grid[calcGridHash(calcGridPos(*(float3*)(p.data + i * p.info.stride), pSysInfo), pSysInfo)].push_back(i);

		long long endTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

		if (endTime - startTime < minTime)
			minTime = endTime - startTime;

//		writeToFile("testFile.raw", grid, pSysInfo.gridSize);
	}
	std::cout << "CPU: Particle count: " << p.info.groupCount << " Time: " << minTime / 1000.0f << std::endl;
	return minTime;
}

int main(int argc, char **argv)
{
	

	Loader loader("laser.mmpld");

	auto pLists = loader.getFrame(20);

	uint3 gridSize;
	gridSize.x = 32;
	gridSize.y = 32;
	gridSize.z = 32;
	PSystemInfo pSysInfo = loader.calcBSystemInfo(gridSize);
	
	/*
	for (int i = 1; i < 11; i++) {
		std::cout << "Grid Size: (" << pSysInfo.gridSize.x << ", " << pSysInfo.gridSize.y << ", " << pSysInfo.gridSize.z << ") Origin: (" << pSysInfo.worldOrigin.x << ", " << pSysInfo.worldOrigin.y << ", " << pSysInfo.worldOrigin.z << ") Cell Size: (" << pSysInfo.cellSize.x << ", " << pSysInfo.cellSize.y << ", " << pSysInfo.cellSize.z << ")" << std::endl;
		benchmarkPListGPU(pLists[0], pSysInfo, 128, 100);
		benchmarkPListCPU(pLists[0], pSysInfo, 100);
		gridSize.x *= 2;
		gridSize.y *= 2;
		gridSize.z *= 2;
		pSysInfo = loader.calcBSystemInfo(gridSize);
	}
	*/
	std::ofstream outputFile("benchmark.csv");
	outputFile << "Partikel Anzahl; GPU; CPU\n";

	
	for (int i = 1; i < 100; i++) {
		ParticleList pList = reduceParticles(pLists[0], 1 - i * 0.001f);
		outputFile << pList.info.groupCount << ";";
		outputFile << benchmarkPListGPU(pList, pSysInfo, 256, 100) << ";";
		outputFile << benchmarkPListCPU(pList, pSysInfo, 100) << "\n";
	}
	outputFile.close();
	return 0;
}
